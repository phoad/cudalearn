#include "hip/hip_runtime.h"
#include "VectorAdd.h"

__global__ void _vectorAdd(float *A, float *B, float *C, unsigned int size, unsigned int column)
{
	unsigned int blk = blockIdx.x * blockDim.x;
	unsigned int dx = threadIdx.x;
	unsigned int dy = threadIdx.y * column;
	unsigned idx = dx + dy + blk;

	//__shared__ float values[1024];

	if (idx < size)
		C[idx] = A[idx] + B[idx];
}


void vectorAdd(float *A, float *B, float *C, unsigned int size)
{
	dim3 threads(32, 32);
	dim3 blocks((size + 1023) / 1024);
	_vectorAdd<<<blocks, threads>>>(A, B, C, size, 32);
}

